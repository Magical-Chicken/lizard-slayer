#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

void *cudaMemcpyToDevice(void *host_data, long size, long item_size) {
    void *device_data = NULL;

    hipMalloc(&device_data, size);
    hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
    return device_data;
}

static void launch_kernel(int Dg, int Db, int Ns, void (*kernel)()) {
    kernel<<<Dg, Db, Ns>>>();
}


static __global__ void aggregate_kernel(void *buf, long count, void *result) {
    /*printf("hello from aggregate kernel\n");*/
    long index = threadIdx.x + blockIdx.x * blockDim.x;

    // FIXME 
    // initial value
    /*type result = 0;*/
    type  *array = (type *)buf;

    if (index < count) {
        atomicAdd((type*)result, array[index]);
    }
}

type aggregate(void *buf, long size, long itemsize, int Dg, int Db, int Ns) {
    printf("hello from cuda\n");
    /*void *device_data = malloc(sizeof(void *));*/
    void *device_data = NULL;
    void *device_result = NULL;

    type result = 0;

    hipMalloc(&device_result, itemsize);
    hipMemcpy(device_result, &result, itemsize, hipMemcpyHostToDevice);

    hipMalloc(&device_data, size);
    hipMemcpy(device_data, buf, size, hipMemcpyHostToDevice);

    /*launch_kernel(Dg, Db, Ns, aggregate_kernel);*/
    aggregate_kernel<<<Dg, Db, Ns>>>(device_data, size / itemsize,
            device_result);

    /*hipDeviceSynchronize();*/
    hipMemcpy(&result, device_result, itemsize, hipMemcpyDeviceToHost);
    hipMemcpy(buf, device_data, size, hipMemcpyDeviceToHost);
    hipFree(device_data);
    hipFree(device_result);
    return result;
}
