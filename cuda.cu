#include <stdio.h>
#include "hip/hip_runtime.h"

static __global__ void testkernel() {
    printf("hello from kernel\n");
}

int test() {
    printf("hello from cuda\n");
    testkernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
