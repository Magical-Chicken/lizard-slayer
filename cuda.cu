#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "hip/hip_runtime.h"

/*
 * Wrapper around hipMemcpy to copy memory to device.
 * returns true on success
 */
bool cudaMemcpyToDevice(void *dst, void *src, long size) {
    return hipMemcpy(dst, src, size, hipMemcpyHostToDevice) == hipSuccess;
}

/*
 * Wrapper around hipMemcpy to copy memory to device.
 * returns true on success
 */
bool cudaMemcpyToHost(void *dst, void *src, long size) {
    return hipMemcpy(dst, src, size, hipMemcpyHostToDevice) == hipSuccess;
}

/*
 * Wrapper around hipMalloc. 
 * returns true on success
 */
bool deviceMalloc(void **dev_ptr, long size) {
    return hipMalloc(dev_ptr, size) == hipSuccess;
}

/*static void launch_kernel(int Dg, int Db, int Ns, void (*kernel)()) {*/
    /*kernel<<<Dg, Db, Ns>>>();*/
/*}*/

static __global__ void kmeans_iteration_kernel(TYPE *centers, TYPE *points, 
        long count, int dim, int k, TYPE *result) {
    long index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < count) {

        int cluster; 
        TYPE shortest = DBL_MAX;
        for (int i = 0; i < k; i++) {
            TYPE mag = 0;

            for (int d = 0; d < dim; d++) {
                TYPE c = points[index * dim + d] - centers[i * dim + d];
                mag = c * c;
            }

            if (mag < shortest)
                cluster = i;
        }
         
        for (int d = 0; d < dim; d++)
            atomicAdd(&result[cluster * dim + d], points[index * dim + d]);
    }
}

static __global__ void aggregate_kernel(void *buf, long count, void *result) {
    /*printf("hello from aggregate kernel\n");*/
    long index = threadIdx.x + blockIdx.x * blockDim.x;

    // FIXME 
    // initial value
    /*TYPE result = 0;*/
    TYPE  *array = (TYPE *)buf;

    if (index < count) {
        atomicAdd((TYPE*)result, array[index]);
    }
}

/*void kmeans_iteration(void *centers, void *points, long size, long itemsize,*/
        /*int k, int dim, int Dg, int Db, int Ns) {*/
    /*void *device_centers = NULL;*/
    /*void *device_points = NULL;*/
    /*void *device_result = NULL;*/

    /*hipMalloc(&device_result, itemsize * k * dim);*/
    /*hipMemcpy(device_result, &result, itemsize, hipMemcpyHostToDevice);*/

    /*hipMalloc(&device_data, size);*/
    /*hipMemcpy(device_data, buf, size, hipMemcpyHostToDevice);*/

    /*kmeans_iteration_kernel<<<Dg, Db, Ns>>>();*/
/*}*/

TYPE aggregate(void *buf, long size, long itemsize, int Dg, int Db, int Ns) {
    printf("hello from cuda\n");
    /*void *device_data = malloc(sizeof(void *));*/
    void *device_data = NULL;
    void *device_result = NULL;

    TYPE result = 0;

    hipMalloc(&device_result, itemsize);
    hipMemcpy(device_result, &result, itemsize, hipMemcpyHostToDevice);

    hipMalloc(&device_data, size);
    hipMemcpy(device_data, buf, size, hipMemcpyHostToDevice);

    /*launch_kernel(Dg, Db, Ns, aggregate_kernel);*/
    aggregate_kernel<<<Dg, Db, Ns>>>(device_data, size / itemsize,
            device_result);

    /*hipDeviceSynchronize();*/
    hipMemcpy(&result, device_result, itemsize, hipMemcpyDeviceToHost);
    hipMemcpy(buf, device_data, size, hipMemcpyDeviceToHost);
    hipFree(device_data);
    hipFree(device_result);
    return result;
}
