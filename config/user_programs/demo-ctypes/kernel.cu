#include "program.h"

void setup_dataset(dataset_t *data, dataset_params_t *params) {
    int i;
    hipMallocManaged(&data->points, params->num_points * sizeof(int*));
    for (i = 0; i < params->num_points; i++)
        hipMallocManaged(&data->points[i], params->dims * sizeof(int));
}

void setup_global_state(global_state_t *state, dataset_params_t *params) {
    int i;
    hipMallocManaged(&state->values, params->dims * sizeof(int));
    for (i = 0; i < params->dims; i++) state->values[i] = 0;
    state->iteration = 0;
}

void setup_aggregation_result(agg_res_t *result, dataset_params_t *params) {
    hipMallocManaged(&result->values, params->dims * sizeof(int));
}

void free_dataset(dataset_t *data, dataset_params_t *params) {
    int i;
    for (i = 0; i < params->num_points; i++)
        hipFree(data->points[i]);
    hipFree(data->points);
}

void free_global_state(global_state_t *state, dataset_params_t *params) {
    hipFree(state->values);
}

void free_aggregation_result(agg_res_t *result, dataset_params_t *params) {
    hipFree(result->values);
}
