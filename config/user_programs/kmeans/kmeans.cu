#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "program.h"

/*
 * Wrapper around hipMemcpy to copy memory to device.
 * returns true on success
 */
bool cudaMemcpyToDevice(void *dst, void *src, long size) {
    return hipMemcpy(dst, src, size, hipMemcpyHostToDevice) == hipSuccess;
}

/*
 * Wrapper around hipMemcpy to copy memory to device.
 * returns true on success
 */
bool cudaMemcpyToHost(void *dst, void *src, long size) {
    return hipMemcpy(dst, src, size, hipMemcpyDeviceToHost) == hipSuccess;
}

/*
 * Wrapper around hipMalloc. 
 * returns true on success
 */
bool deviceMalloc(void **dev_ptr, long size) {
    return hipMalloc(dev_ptr, size) == hipSuccess;
}

bool deviceFree(void *dev_ptr) {
    return hipFree(dev_ptr) == hipSuccess;
}

static __global__ void kmeans_iteration_kernel(double *centers, double *points,
        double *partial_results, int *count_results, long count, int dim, int k) {
    long index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < count) {
        int cluster = -1; 
        double shortest = DBL_MAX;
        for (int i = 0; i < k; i++) {
            TYPE mag = 0;

            for (int d = 0; d < dim; d++) {
                double c = points[index * dim + d] - centers[i * dim + d];
                mag += c * c;
            }
            /*printf("mag %lf\n", mag);*/

            if (mag < shortest) {
                shortest = mag;
                cluster = i;
            }
        }
        /*printf("kernel: point %lf\n", points[index]);*/
        /*printf("kernel: cluster %i\n", cluster);*/
         
        for (int d = 0; d < dim; d++) {
            atomicAdd(&partial_results[cluster * dim + d], points[index * dim + d]);
            atomicAdd(&count_results[cluster], 1);
            /*printf("results: %lf\n", results[cluster *dim+d]);*/
        }
    }
}

static __global__ void aggregate_kernel(void *buf, long count, void *result) {
    /*printf("hello from aggregate kernel\n");*/
    long index = threadIdx.x + blockIdx.x * blockDim.x;

    // FIXME 
    // initial value
    /*TYPE result = 0;*/
    TYPE  *array = (TYPE *)buf;

    if (index < count) {
        atomicAdd((TYPE*)result, array[index]);
    }
}

void kmeans_iteration(double *centers, double *dev_points, double *dev_partial_results, 
        int *dev_count_results, long size, long itemsize, int k, int dim, int Dg, int Db, int Ns) {
    double *dev_centers = NULL;

    hipMalloc(&dev_centers, itemsize * k * dim);
    hipMemcpy(dev_centers, centers, itemsize * k * dim, hipMemcpyHostToDevice);

    /*printf("count: %i\n", size / itemsize/ dim);*/
    /*for (int i = 0; i < 4; i++) */
        /*printf("%lf\n", centers[i]);*/

    kmeans_iteration_kernel<<<Dg, Db, Ns>>>(dev_centers, dev_points,
            dev_partial_results, dev_count_results, size / itemsize / dim, dim, k);

    hipFree(dev_centers);
}

TYPE aggregate(void *buf, long size, long itemsize, int Dg, int Db, int Ns) {
    printf("hello from cuda\n");
    /*void *device_data = malloc(sizeof(void *));*/
    void *device_data = NULL;
    void *device_result = NULL;

    TYPE result = 0;

    hipMalloc(&device_result, itemsize);
    hipMemcpy(device_result, &result, itemsize, hipMemcpyHostToDevice);

    hipMalloc(&device_data, size);
    hipMemcpy(device_data, buf, size, hipMemcpyHostToDevice);

    /*launch_kernel(Dg, Db, Ns, aggregate_kernel);*/
    aggregate_kernel<<<Dg, Db, Ns>>>(device_data, size / itemsize,
            device_result);

    /*hipDeviceSynchronize();*/
    hipMemcpy(&result, device_result, itemsize, hipMemcpyDeviceToHost);
    hipMemcpy(buf, device_data, size, hipMemcpyDeviceToHost);
    hipFree(device_data);
    hipFree(device_result);
    return result;
}
