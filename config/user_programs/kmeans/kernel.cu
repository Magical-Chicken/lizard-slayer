#include "program.h"

void run_iteration(int block_size, global_params_t *params,
        dataset_t *data, global_state_t *state, agg_res_t *result) {
}

void setup_dataset(dataset_t *data, global_params_t *params) {
    int i;
    hipMallocManaged(&data->points, data->num_points * sizeof(float*));
    for (i = 0; i < data->num_points; i++)
        hipMallocManaged(&data->points[i], params->dims * sizeof(float));
}

void setup_global_state(global_state_t *state, global_params_t *params) {
    int i;
    hipMallocManaged(
        &state->centroids, params->num_centroids * sizeof(float*));
    for (i = 0; i < params->num_centroids; i++)
        hipMallocManaged(&state->centroids[i], params->dims * sizeof(float));
}

void setup_aggregation_result(agg_res_t *result, global_params_t *params) {
    int i;
    hipMallocManaged(
        &result->centroid_updates, params->num_centroids * sizeof(float*));
    for (i = 0; i < params->num_centroids; i++)
        hipMallocManaged(
            &result->centroid_updates[i], params->dims * sizeof(float));
    hipMallocManaged(
        &result->update_counts, params->num_centroids * sizeof(int));
}

void free_dataset(dataset_t *data, global_params_t *params) {
    int i;
    for (i = 0; i < data->num_points; i++)
        hipFree(data->points[i]);
    hipFree(data->points);
}

void free_global_state(global_state_t *state, global_params_t *params) {
    int i;
    for (i = 0; i < params->num_centroids; i++)
        hipFree(state->centroids[i]);
    hipFree(state->centroids);
}

void free_aggregation_result(agg_res_t *result, global_params_t *params) {
    int i;
    for (i = 0; i < params->num_centroids; i++)
        hipFree(result->centroid_updates[i]);
    hipFree(result->centroid_updates);
    hipFree(result->update_counts);
}
